
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define PerThread 1024*16//每个线程计算多少个i
#define N 64*256*1024*16//积分计算PI总共划分为这么多项相加
#define BlockNum 64 //block的数量
#define ThreadNum 256 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{
    __shared__ double cache[ThreadNum];//每个block共享一个shared memory.
    int cacheIdx=threadIdx.x;
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;
    int begin=tid*PerThread+1;
    int end=begin+PerThread;
    double temp=0;
    int flag=1;
    for(int i=begin;i<end;i++){
        temp+=flag*(1.0/(2*i-1));
        flag=flag*(-1);
    }
    cache[cacheIdx]=temp;
    __syncthreads();

    int i=blockDim.x/2;
    while(i!=0){
        if(cacheIdx<i) cache[cacheIdx]+=cache[cacheIdx+i];
        __syncthreads();
        i=i/2;
    }

    if(cacheIdx==0){
        Gpu_list[blockIdx.x]=cache[0];
    }
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);
    // printf("go to GPU\n");
    double begin=clock();
    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=4*outcome;
    double end=clock();
    printf("CudaShareMemoryPI-2: N=%d, PI value=%.10f, Using time =%.10f\n",N,outcome,(end-begin)/(CLOCKS_PER_SEC));

    
}