
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>

#define PerThread 1024*4*8//每个线程计算多少个i
#define N 64*256*1024*4//积分计算PI总共划分为这么多项相加
#define BlockNum 32 //block的数量
#define ThreadNum 64 //每个block中threads的数量

__global__ void Gpu_calPI(double* Gpu_list)
{   //核函数
    int tid=blockIdx.x*blockDim.x*blockDim.y+threadIdx.x;//计算线程号
    int begin=tid*PerThread;
    int end=begin+PerThread-1;//计算每个线程的工作范围
    double temp=0;
    for(int i=begin;i<end;i++){
        temp+=4.0/(1+((i+0.5)/(N))*((i+0.5)/(N)));
    }
    Gpu_list[tid]=temp;//存入计算结果
}

int main(void)
{
    double * cpu_list;
    double * Gpu_list;
    double outcome=0;
    cpu_list=(double*)malloc(sizeof(double)*BlockNum*ThreadNum);
    hipMalloc((void**)&Gpu_list,sizeof(double)*BlockNum*ThreadNum);
    // dim3 blocksize=dim3(1,ThreadNum);
    // dim3 gridsize=dim3(1,BlockNum);
    double begin=clock();
    Gpu_calPI<<<BlockNum,ThreadNum>>>(Gpu_list);

    hipMemcpy(cpu_list,Gpu_list,sizeof(double)*BlockNum*ThreadNum,hipMemcpyDeviceToHost);
    for(int i=0;i<BlockNum*ThreadNum;i++){
        outcome+=cpu_list[i];
    }
    outcome=outcome/(N);
    double end=clock();
    printf("CudaPI-1: N=%d, PI value=%.10f, Using time =%.10f\n",N,outcome,(end-begin)/(CLOCKS_PER_SEC));

    
}